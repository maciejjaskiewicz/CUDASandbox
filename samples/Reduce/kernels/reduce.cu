#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <>
#include <CUDASandbox/gpu_assert.hpp>
#include <cstdint>
#include <cstdio>
#include <vector>

#include "reduce.cuh"

// high warp divergence!!!
__global__ void reduce_neighbored_pairs(int* input, int* result, uint32_t size)
{
	const int tid = threadIdx.x;
	const int gid = blockDim.x * blockIdx.x + tid;

	if(gid > size) return;
	
	for(auto offset = 1; offset <= blockDim.x / 2; offset *= 2)
	{
		if(tid % (2 * offset) == 0)
		{
			input[gid] += input[gid + offset];
		}

		__syncthreads();
	}

	if(tid == 0)
	{
		result[blockIdx.x] = input[gid];
	}
}

template<typename T>
void init_device(T*& d_data, T*& d_result, const std::vector<T>& data, 
	const std::size_t result_size, metric& metric)
{
	metric.start(metric_type::MEMORY_TRANSFER);

	const uint32_t data_byte_size = data.size() * sizeof(T);
	const uint32_t result_byte_size = result_size * sizeof(T);

	GPU_ERR_CHECK(hipMalloc(reinterpret_cast<void**>(&d_data), data_byte_size));
	GPU_ERR_CHECK(hipMalloc(reinterpret_cast<void**>(&d_result), result_byte_size));

	GPU_ERR_CHECK(hipMemcpy(d_data, data.data(), data_byte_size, hipMemcpyHostToDevice));
	GPU_ERR_CHECK(hipMemset(d_result, 0, result_byte_size));

	metric.stop(metric_type::MEMORY_TRANSFER);
}

template<typename T>
T fetch_device_result(const T* d_result, const std::size_t size)
{
	const uint32_t result_byte_size = size * sizeof(T);

	std::vector<T> h_result(size);
	GPU_ERR_CHECK(hipMemcpy(h_result.data(), d_result, result_byte_size, hipMemcpyDeviceToHost));

	T gpu_result = 0;

	for(const auto& val : h_result)
	{
		gpu_result += val;
	}

	return gpu_result;
}

template<typename T>
metric reduce_gpu::reduce_neighbored(const std::vector<T>& data, const uint16_t block_size)
{
	T* d_data;
	T* d_result;
	metric metric(data.size());

	const dim3 block(block_size);
	const dim3 grid(data.size() / block_size);

	init_device(d_data, d_result, data, grid.x, metric);

	metric.start(metric_type::CALCULATION);
	reduce_neighbored_pairs<<<grid, block>>>(d_data, d_result, data.size());
	GPU_ERR_CHECK(hipDeviceSynchronize());
	metric.stop(metric_type::CALCULATION);

	T gpu_result = fetch_device_result(d_result, grid.x);
	printf("GPU Result: %d\n", gpu_result);

	GPU_ERR_CHECK(hipFree(d_data));
	GPU_ERR_CHECK(hipFree(d_result));

	return metric;
}

// Explicit instantiations
template metric reduce_gpu::reduce_neighbored(const std::vector<int>& data, uint16_t block_size);
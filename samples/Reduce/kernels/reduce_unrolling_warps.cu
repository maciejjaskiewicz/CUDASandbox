#include "hip/hip_runtime.h"
#include <>

#include "reduce.cuh"
#include "reduce_helpers.cuh"

__global__ void reduce_unrolling_warps_kernel(int* input, int* result, uint32_t size)
{
    const int tid = threadIdx.x;
    int* input_local = input + blockDim.x * blockIdx.x;
    
    for(auto offset = blockDim.x / 2; offset >= 64; offset /= 2)
    {
        if(tid < offset)
        {
            input_local[tid] += input_local[tid + offset];
        }

        __syncthreads();
    }

    if(tid < 32)
    {
        volatile int* vsmem = input_local;
        vsmem[tid] += vsmem[tid + 32];
        vsmem[tid] += vsmem[tid + 16];
        vsmem[tid] += vsmem[tid + 8];
        vsmem[tid] += vsmem[tid + 4];
        vsmem[tid] += vsmem[tid + 2];
        vsmem[tid] += vsmem[tid + 1];
    }

    if(tid == 0)
    {
        result[blockIdx.x] = input_local[0];
    }
}

template<typename T>
metric_with_result<T> reduce_gpu::reduce_unrolling_warps(const std::vector<T>& data, const uint16_t block_size)
{
    T* d_data;
    T* d_result;
    metric_with_result<T> metric(data.size());

    const dim3 block(block_size);
    const dim3 grid(data.size() / block_size);

    init_device(d_data, d_result, data, grid.x, metric);

    metric.start(metric_type::CALCULATION);
    reduce_unrolling_warps_kernel<<<grid, block>>>(d_data, d_result, data.size());
    GPU_ERR_CHECK(hipDeviceSynchronize());
    metric.stop(metric_type::CALCULATION);

    T gpu_result = fetch_device_result(d_result, grid.x);
    metric.set_result(gpu_result);

    GPU_ERR_CHECK(hipDeviceReset());

    return metric;
}

// Explicit instantiations
template metric_with_result<int> reduce_gpu::reduce_unrolling_warps(const std::vector<int>& data, uint16_t block_size);
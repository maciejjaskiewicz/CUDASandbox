#include "hip/hip_runtime.h"
#include <>

#include "reduce.cuh"
#include "reduce_helpers.cuh"

// high warp divergence!!!
__global__ void reduce_neighbored_pairs_imp(int* input, int* result, uint32_t size)
{
	const int tid = threadIdx.x;
	const int gid = blockDim.x * blockIdx.x + tid;
	int* input_local = input + blockDim.x * blockIdx.x;

	if(gid > size) return;
	
	for(auto offset = 1; offset <= blockDim.x / 2; offset *= 2)
	{
		int index = 2 * offset * tid;

		if(index < blockDim.x)
		{
			input_local[index] += input_local[index + offset];
		}

		__syncthreads();
	}

	if(tid == 0)
	{
		result[blockIdx.x] = input[gid];
	}
}

template<typename T>
metric_with_result<T> reduce_gpu::reduce_neighbored_imp(const std::vector<T>& data, const uint16_t block_size)
{
	T* d_data;
	T* d_result;
	metric_with_result<T> metric(data.size());

	const dim3 block(block_size);
	const dim3 grid(data.size() / block_size);

	init_device(d_data, d_result, data, grid.x, metric);

	metric.start(metric_type::CALCULATION);
	reduce_neighbored_pairs_imp<<<grid, block>>>(d_data, d_result, data.size());
	GPU_ERR_CHECK(hipDeviceSynchronize());
	metric.stop(metric_type::CALCULATION);

	T gpu_result = fetch_device_result(d_result, grid.x);
	metric.set_result(gpu_result);

	GPU_ERR_CHECK(hipDeviceReset());

	return metric;
}

// Explicit instantiations
template metric_with_result<int> reduce_gpu::reduce_neighbored_imp(const std::vector<int>& data, uint16_t block_size);